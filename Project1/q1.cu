#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<>

__global__ void multiplyKernel_rowwise(int* a, int* b, int* c, int wa, int wb) {
	int ridA = threadIdx.x;
	int sum;

	for (int cidB = 0; cidB < wb; cidB++) {
		sum = 0;
		for (int k = 0; k < wa; k++) {
			sum += (a[ridA * wa + k] * b[k * wb + cidB]);
		}
		c[ridA * wb + cidB] = sum;
	}
}

__global__ void multiplyKernel_columnwise(int* a, int* b, int* c, int ha, int wa) {
	int cidB = threadIdx.x;
	int wb = blockDim.x;
	int sum, k;
	for (int ridA = 0; ridA < ha; ridA++) {
		sum = 0;
		for (k = 0; k < wa; k++)
			sum += (a[ridA * wa + k] * b[k * wb + cidB]);
		c[ridA * wb + cidB] = sum;
	}
}

__global__ void multiplyKernel_elementwise(int* a, int* b, int* c, int wa) {
	int ridA = threadIdx.y;
	int cidB = threadIdx.x;
	int wb = blockDim.x;
	int sum = 0, k;

	for (k = 0; k < wa; k++)
		sum += (a[ridA * wa + k] * b[k * wb + cidB]);
	
	c[ridA * wb + cidB] = sum;
}
int main() {
	int* a, * b, * c;
	int* da, * db, * dc;
	int rowsA, colsA, rowsB, colsB;

	printf("Enter rows and columns of matrix A:");
	scanf("%d %d", &rowsA, &colsA);

	printf("Enter rows and columns of matrix B:");
	scanf("%d %d", &rowsB, &colsB);

	if (colsA != rowsB) {
		printf("Matrix multiplication not possible");
		return 1;
	}

	int sizeA = sizeof(int) * rowsA * colsA;
	int sizeB = sizeof(int) * rowsB * colsB;
	int sizeC = sizeof(int) * rowsA * colsB;

	a = (int*)malloc(sizeA);
	b = (int*)malloc(sizeB);
	c = (int*)malloc(sizeC);

	printf("Enter the elements of matrix A: ");
	for (int i = 0; i < rowsA * colsA; i++)
		scanf("%d", &a[i]);

	printf("Enter the elements of matrix B: ");
	for (int i = 0; i < rowsB * colsB; i++)
		scanf("%d", &b[i]);

	hipMalloc((void**)&da, sizeA);
	hipMalloc((void**)&db, sizeB);
	hipMalloc((void**)&dc, sizeC);

	hipMemcpy(da, a, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(db, b, sizeB, hipMemcpyHostToDevice);

	//multiplyKernel_rowwise << <1, rowsA >> > (da, db, dc, colsA, colsB);
	//multiplyKernel_columnwise << < 1, colsB >> > (da, db, dc, rowsA, colsA);
	multiplyKernel_elementwise << <dim3(1, 1), dim3(colsB, rowsA) >> > (da, db, dc, colsA);
	hipMemcpy(c, dc, sizeC, hipMemcpyDeviceToHost);
	
	printf("resultant matrix: \n");
	for (int i = 0; i < rowsA; i++) {
		for (int j = 0; j < colsB; j++)
			printf("%d\t", c[i * colsB + j]);
		printf("\n");
	}

	hipFree(da);
	hipFree(db);
	hipFree(dc);
	
	return 0;
}